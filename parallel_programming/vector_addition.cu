#include "hip/hip_runtime.h"
// MP 1
#include <wb.h>

__global__ void vecAdd(float *in1, float *in2, float *out, int len) {
  //@@ Insert code to implement vector addition here
  int i = blockIdx.x*blockDim.x + threadIdx.x;
  if(i<len)
    out[i] = in1[i]+in2[i];
}

#define ThreadsPerBlock 16

int main(int argc, char **argv) {
  wbArg_t args;
  int inputLength;
  float *hostInput1;
  float *hostInput2;
  float *hostOutput;
  float *deviceInput1;
  float *deviceInput2;
  float *deviceOutput;

  args = wbArg_read(argc, argv);

  wbTime_start(Generic, "Importing data and creating memory on host");
  hostInput1 = ( float * )wbImport(wbArg_getInputFile(args, 0), &inputLength);
  hostInput2 = ( float * )wbImport(wbArg_getInputFile(args, 1), &inputLength);
  hostOutput = ( float * )malloc(inputLength * sizeof(float));
  wbTime_stop(Generic, "Importing data and creating memory on host");

  wbLog(TRACE, "The input length is ", inputLength);

  hipError_t err;
  wbTime_start(GPU, "Allocating GPU memory.");
  //@@ Allocate GPU memory here
  err = hipMalloc((void**)&deviceInput1, inputLength*sizeof(float));
  err = hipMalloc((void**)&deviceInput2, inputLength*sizeof(float));
  err = hipMalloc((void**)&deviceOutput, inputLength*sizeof(float));

  wbTime_stop(GPU, "Allocating GPU memory.");

  wbTime_start(GPU, "Copying input memory to the GPU.");
  //@@ Copy memory to the GPU here
  err = hipMemcpy(deviceInput1, hostInput1, sizeof(float)*inputLength, hipMemcpyHostToDevice);
  err = hipMemcpy(deviceInput2, hostInput2, sizeof(float)*inputLength, hipMemcpyHostToDevice);

  wbTime_stop(GPU, "Copying input memory to the GPU.");

  //@@ Initialize the grid and block dimensions here
  dim3 numBlocks(((inputLength-1)/ThreadsPerBlock)+1,1,1);
  dim3 numThreads(ThreadsPerBlock,1,1);

  wbTime_start(Compute, "Performing CUDA computation");
  //@@ Launch the GPU Kernel here
  vecAdd<<<numBlocks, numThreads>>>(deviceInput1, deviceInput2, deviceOutput, inputLength);

  hipDeviceSynchronize();
  wbTime_stop(Compute, "Performing CUDA computation");

  wbTime_start(Copy, "Copying output memory to the CPU");
  //@@ Copy the GPU memory back to the CPU here
  err = hipMemcpy(hostOutput, deviceOutput, inputLength*sizeof(float), hipMemcpyDeviceToHost);

  wbTime_stop(Copy, "Copying output memory to the CPU");

  wbTime_start(GPU, "Freeing GPU Memory");
  //@@ Free the GPU memory here
  err = hipFree(deviceInput1);
  err = hipFree(deviceInput2);
  err = hipFree(deviceOutput);

  wbTime_stop(GPU, "Freeing GPU Memory");

  wbSolution(args, hostOutput, inputLength);

  free(hostInput1);
  free(hostInput2);
  free(hostOutput);

  return 0;
}

